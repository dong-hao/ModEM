#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#if defined(FG)
#include <nccl.h>
#endif

// simple kernel function that converts double vectors to single
__global__ void real64to32(const double *in, float *out, const int N)
{
    // a position every 64 bits
    // int pos = blockDim.x * blockIdx.x + threadIdx.x;
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	float sing = __double2float_rn(in[pos]);
	out[pos] = sing;
    }
}

// simple kernel function that converts single vectors to double
__global__ void real32to64(const uint32_t *in, uint64_t *out, const int N)
{
    // a position every 32 bits
    // int pos = blockDim.x * blockIdx.x + threadIdx.x;
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	// last sign bit (1)
	uint64_t s = in[pos] & 0x80000000;
	// exponent bits (8)
	uint64_t e = ((in[pos] & 0x7f800000) >> 23);
	e = e + 896;
	// mantissa bits (23)
	uint64_t m = in[pos] & 0x7fffff;
	// double through bitwise or
	uint64_t doub = (s<<32) | (e<<52) | (m<<29);
        // a new position every 64 bits
	out[pos] = doub;
    }
 }

__global__ void hada_real(const double *ina, const double *inb, double *out, const int N)
{
    //hadamard multiplication, as real 
    // c = a * b
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	out[pos] = ina[pos]*inb[pos];
    }
 }

__global__ void hada_cmplx(const double *ina, const double *inb, double *out, const int N)
{
    //hadamard multiplication, as complex
    //(a + bi) * (c + di) = (ac -bd) + (ad + bc)i
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
        if ( pos%2 == 0) // 0 2 4 
        {
	        out[pos] = ina[pos]*inb[pos]-ina[pos+1]*inb[pos+1];
        }
        else // 1 3 5 
        {
	        out[pos] = ina[pos-1]*inb[pos]+ina[pos]*inb[pos-1];
        }
    }
 }

__global__ void xpby_real(const double *x, const double b, double *y, const int N)
{
    // CUDA kernel implementing xpby:
    // y = x + b*y
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	        y[pos] = x[pos]+b*y[pos];
    }
}

__global__ void xpby_cmplx(const double *x, double br, double bi, double *y, const int N)
{
    // CUDA kernel implementing xpby:
    // y = x + b*y
    // complex makes this a little tricky - need a temp variable
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // lower 64bit --> real
	    double lower = x[pos] + br*y[pos] - bi*y[pos+1];
        // higher 64bit --> imag
	    y[pos+1] = x[pos+1] + br*y[pos+1] + bi*y[pos];
        // and copy them back
        y[pos] = lower;
    }
}

__global__ void p_update_real(const double *v, const double *r, const double beta, const double omega, double *p, const int N)
{
    // CUDA kernel implementing p update:
    // p = r + beta * (p - omega*v)
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    if (pos<N) 
    {  
	    p[pos] = r[pos] + beta*(p[pos] - omega*v[pos]);
    }
}

__global__ void p_update_cmplx(const double *r, const double *v, const double br, const double bi, const double wr, const double wi, double *p, const int N)
{
    // CUDA kernel implementing p update:
    // p = r + beta * (p - omega*v)
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // lower 64bit --> real p - omega*v
	    double lower = p[pos] - wr*v[pos] + wi*v[pos+1];
        // higher 64bit --> imag p - omega*v
	    double higher = p[pos+1] - wr*v[pos+1] - wi*v[pos];
        // lower 64bit --> real r + beta * (p - omega*v)
	    p[pos] = r[pos] + br*lower - bi*higher;
        // higher 64bit --> imag r + beta * (p - omega*v)
	    p[pos+1] = r[pos+1] + br*higher + bi*lower;
    }
}

__global__ void x_update_cmplx(const double *p, const double *s, const double ar, const double ai, const double wr, const double wi, double *x, const int N)
{
    // CUDA kernel implementing p update:
    int pos = ((blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x)*2;
    if (pos<N) 
    {  
        // x = x + alpha * ph + omega * sh
        // lower 64bit --> real x + alpha*ph
	    x[pos] = x[pos] + ar*p[pos] - ai*p[pos+1] + wr*s[pos] - wi*s[pos];
        // higher 64bit --> imag x + alpha*ph
	    x[pos+1]= x[pos+1] + ar*p[pos+1] + ai*p[pos] + wr*s[pos+1] + wi*s[pos];
    }
}

__global__ void reduce_real(double *a, double s, const int N)
{
    // CUDA kernel implementing reduce s = sum(a)
    int pos = (blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
    int number_of_threads = (blockDim.x*blockDim.y);
    int step = 1; //initial step
    while (number_of_threads > 0)
    {
        if (pos < number_of_threads) 
        {
            const int left = pos * step*2; 
            const int right = left + step;
            a[left] += a[right];
        }
        step <<= 1;
        number_of_threads >>= 1;
        __syncthreads();
    }
}

// function to wait for sometime
void sleep(int seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000 * seconds;
    // Storing start time
    clock_t start_time = clock();
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

// function called from main fortran program
// single to double conversion
extern "C" void kernelc_s2d(const uint32_t *a_d, uint64_t *b_d, int Np)
{
    //uint32_t  *a_d;  // declare GPU vector double (but stored as uint)
    //uint64_t  *b_d;  // declare GPU vector float
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    real32to64<<< grids, blocks >>>( a_d, b_d, N);

    return;
}

// function called from main fortran program
// double to single conversion
extern "C" void kernelc_d2s(const double *a_d, float *b_d, int Np)
{
    //double  *a_d;  // declare GPU vector double 
    //float  *b_d;  // declare GPU vector float
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    real64to32<<< grids, blocks >>>( a_d, b_d, N);

    return;
}

// function called from main fortran program
// hadamard multiply real version
extern "C" void kernelc_hadar(const double *a_d, const double *b_d, double *c_d,
		int Np, hipStream_t cstream)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
    //double  *c_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    hada_real<<< grids, blocks, 0, cstream >>>( a_d, b_d, c_d, N);

    return;
}

// function called from main fortran program
// hadamard multiply complex version
extern "C" void kernelc_hadac(const double *a_d, const double *b_d, double *c_d,
		int Np, hipStream_t cstream)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
    //double  *c_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = Np*2;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // call function on GPU
    hada_cmplx<<< grids, blocks, 0, cstream >>>( a_d, b_d, c_d, N);

    return;
}

// function called from main fortran program
// xpby complex version
extern "C" void kernelc_xpbyc(const double *x_d, double _Complex b, double *y_d,
		int Np, hipStream_t cstream)
{
    //double  *x_d;  // declare GPU vector double 
    //double  *y_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double br = creal(b);
    // get lower 64 bit
    double bi = cimag(b);
    // call function on GPU
    xpby_cmplx<<< grids, blocks, 0, cstream>>>( x_d, br, bi, y_d, N);

    return;
}

// function called from main fortran program
// update p complex version
extern "C" void kernelc_update_pc(const double *r_d, const double *v_d, double _Complex beta, double _Complex omega, double *p_d, int Np, hipStream_t cstream)
{
    //double  *a_d;  // declare GPU vector double 
    //double  *b_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double br = creal(beta);
    // get lower 64 bit
    double bi = cimag(beta);
    // get upper 64 bit
    double wr = creal(omega);
    // get lower 64 bit
    double wi = cimag(omega);
    // call function on GPU
    p_update_cmplx<<< grids, blocks, 0, cstream >>>( r_d, v_d, br, bi, wr, wi, p_d, N);

    return;
}

// function called from main fortran program
// update x complex version
extern "C" void kernelc_update_xc(const double *ph_d, const double *sh_d,
		double _Complex alpha, double _Complex omega, double *x_d,
		int Np, hipStream_t cstream)
{
    //double  *ph_d;  // declare GPU vector double 
    //double  *sh_d;  // declare GPU vector double
    //double  *x_d;  // declare GPU vector double
           
    int ngrid;              // number of grid
    int N;                  // length of N 

    N = 2*Np;
    ngrid = N/256 + 1;
    dim3 grids(ngrid,1,1);
    dim3 blocks(32,8,1);
    // get upper 64 bit
    double ar = creal(alpha);
    // get lower 64 bit
    double ai = cimag(alpha);
    // get upper 64 bit
    double wr = creal(omega);
    // get lower 64 bit
    double wi = cimag(omega);
    // call function on GPU
    x_update_cmplx<<< grids, blocks, 0, cstream >>>( ph_d, sh_d, ar, ai, wr, wi, x_d, N);

    return;
}

#if defined(FG)
// This function is copied from nccl
static __inline__ int ncclTypeSize(ncclDataType_t type) {
  switch (type) {
    case ncclInt8:
    case ncclUint8:
      return 1;
    case ncclFloat16:
      return 2;
    case ncclInt32:
    case ncclUint32:
    case ncclFloat32:
      return 4;
    case ncclInt64:
    case ncclUint64:
    case ncclFloat64:
      return 8;
    default:
      return -1;
  }
}
// a homebrew allgatherv method by stitching the basic nccl apis togather  
extern "C" ncclResult_t ncclAllGatherV(void *sendbuff, size_t sendcount,
	ncclDataType_t senddatatype, void *recvbuff,
	size_t recvcounts[], size_t recvdispls[], 
	int root, int n, ncclComm_t comm, hipStream_t stream)
{
    int rank_nccl;
    int size_nccl;
    hipError_t cuErr;
    ncclResult_t Err;
    // need to check the nccl interface here
    ncclCommUserRank(comm, &rank_nccl);
    ncclCommCount(comm, &size_nccl);
    // for debug
    // printf("#C rank = %i, size = %i \n", rank_nccl, size_nccl);
    // for debug
    // printf("#C rank = %i, sizes = %ld \n", rank_nccl, recvcounts[rank_nccl]);
    // for debug
    // printf("#C rank = %i, displs = %ld \n",rank_nccl, recvdispls[rank_nccl]);
    // firstly gather to root
    // send to root (if you are not)
    if(rank_nccl!=root) {
	// non-root send the data
        // printf("#C sending data to %i @ %i\n", 0, rank_nccl);
        Err = ncclSend(sendbuff, sendcount, senddatatype,
        	root, comm, stream);
        if(Err){
            return Err;
	}
    }
    else {
        // root firstly copy sendbuff to recvbuff 
        size_t self_displ = recvdispls[rank_nccl];
        cuErr = hipMemcpyAsync(static_cast<std::byte*>(recvbuff) + 
	    ncclTypeSize(senddatatype) * self_displ,
            sendbuff, sendcount * ncclTypeSize(senddatatype),
            hipMemcpyDeviceToDevice, stream);
	if (cuErr != hipSuccess) return ncclSystemError;
	// then recieve the data from other processes
        for(int i=0;i<size_nccl;++i){
            if(i == root) continue; //skip the root
            // printf("#C waiting for data from %i @ %i\n", i, rank_nccl);
            Err=ncclRecv(static_cast<std::byte*>(recvbuff)+
	        ncclTypeSize(senddatatype)*recvdispls[i],
                recvcounts[i], senddatatype, i, comm, stream);
            if(Err){
        	 return Err;
	    }
	}
    }
    // now broadcast to all
    size_t total = 0;
    for (int i = 0; i < size_nccl; i++){
        total += recvcounts[i];
    }
    // for debug
    // now broadcast the full vector
    // printf("#C now broadcasting @ %i\n", rank_nccl);
    Err=ncclBcast(recvbuff, total, senddatatype, 0, comm, stream);
    if(Err){
        return Err;
    }
    return ncclSuccess;
}

// a homebrew allgatherv method by stitching the basic nccl apis togather
extern "C" ncclResult_t ncclAllGatherV2(void *sendbuff, size_t sendcount,
	ncclDataType_t senddatatype, void *recvbuff,
        size_t recvcounts[], size_t recvdispls[],
        int root, int n, ncclComm_t comm, hipStream_t stream)
{
    int rank_nccl;
    int size_nccl;
    hipError_t cuErr;
    ncclResult_t Err;

    // Get the rank and size of the communicator
    Err = ncclCommUserRank(comm, &rank_nccl);
    if (Err != ncclSuccess) return Err;
    Err = ncclCommCount(comm, &size_nccl);
    if (Err != ncclSuccess) return Err;

    // firstly everyone copies the local sendbuff to recvbuff
    size_t size_byte  = sendcount * ncclTypeSize(senddatatype);
    size_t displ_byte = recvdispls[rank_nccl] * ncclTypeSize(senddatatype);
    cuErr = hipMemcpyAsync(static_cast<std::byte*>(recvbuff) + displ_byte,
            sendbuff, size_byte, hipMemcpyDeviceToDevice,stream);
    if (cuErr != hipSuccess) {
        printf("Failed to copy the device Mem: %s\n",
           hipGetErrorString(cuErr));
        return ncclSystemError;
    }

    // Perform the allgather operation using a ring communication pattern
    // find the right and left neighbour
    int left = (rank_nccl - 1 + size_nccl) % size_nccl;
    int right = (rank_nccl + 1) % size_nccl;

    // we perform an in-place send and recv to avoid frequent alloc/dealloc
    // operations
    // For each rank, we'll send and receive data
    // except the local one (as we already have that)
    for (int i = 1; i < size_nccl; ++i) {
        // Synchronize the gather operation
        Err = ncclGroupStart();
        if (Err != ncclSuccess) return Err;
        // always Send data to the right neighbor, use the size
	// and displs from the last iteration
        Err = ncclSend(static_cast<std::byte*>(recvbuff) + displ_byte,
                      size_byte, senddatatype, right, comm, stream);
        if (Err) {
            return Err;
        }
	// size and displs to receive from the rank to the left
	int next = (i + rank_nccl - 2 + size_nccl) % size_nccl;
	size_byte = recvcounts[next] * ncclTypeSize(senddatatype);
	displ_byte = recvdispls[next] * ncclTypeSize(senddatatype);
        // always Receive data from the left neighbor
        Err = ncclRecv(static_cast<std::byte*>(recvbuff) + displ_byte,
                      size_byte, senddatatype, left, comm, stream);
        if (Err) {
            return Err;
        }
        // Synchronize the gather operation
        Err = ncclGroupEnd();
        if (Err != ncclSuccess) return Err;
    }

    return ncclSuccess;
}

// function called from main fortran program
#endif

extern "C" int cf_resetFlag(int dev_idx)
{
    hipError_t err;
    unsigned int flag = 0;
    err = hipGetDeviceFlags(&flag);
    if (err != hipSuccess)
    {
        printf("Failed to get the device flags %i: %s\n", dev_idx, 
           hipGetErrorString(err));
    }
    // printf("Dev = %i, flag = %i \n", dev_idx, flag);
    if (flag == hipDeviceScheduleYield)
    {
        err = hipSetDeviceFlags(hipDeviceScheduleSpin);
        if (err != hipSuccess)
        {
            printf("Failed to set the device flag %i: %s\n", dev_idx, 
            hipGetErrorString(err));
            return 1;
        }
    }
    else
    {
        sleep(0.1);
        err = hipSetDeviceFlags(hipDeviceScheduleYield);
        if (err != hipSuccess)
        {
            printf("Failed to set the device flag %i: %s\n", dev_idx, 
            hipGetErrorString(err));
            return 1;
        }
    }
    return 0;
}

// function called from main fortran program 
extern "C" int cf_hookDev(int dev_idx)
{
    hipDeviceProp_t dev_prop;
    hipError_t err;
    size_t freeBytes, totalBytes, usedBytes;
    int nDevices;
    unsigned int flag = 0;
    double memusage;
     
    // unsigned int i;
    err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) 
    {
        printf("Error: Failed to inquire the number of devices %s\n", hipGetErrorString(err));
        return 1;
    }
    // see the number of devices
    if (dev_idx + 1 > nDevices) 
    { 
        printf("Error: device idx out of range! \n");
        printf(" inquired device idx = %i, nDevices = %u \n", dev_idx, nDevices);
        goto Error;
    }
    err = hipGetDeviceProperties(&dev_prop, dev_idx);
    //printf("Device Idx: %d, name: %s \n", dev_idx, dev_prop.name);
    //printf("  Memory Clock Rate (KHz): %d\n",
    //         dev_prop.memoryClockRate);
    //printf("  Memory Bus Width (bits): %d\n",
    //         dev_prop.memoryBusWidth);
    //printf("  Peak Memory Bandwidth (GB/s): %f\n",
    //         2.0*dev_prop.memoryClockRate*(dev_prop.memoryBusWidth/8)/1.0e6);
    //printf("  PCI bus ID: %i\n",
    //         dev_prop.pciBusID);
    //printf("  is integrated GPU : %i\n",
    //         dev_prop.integrated);
    // now try to get 
    if (dev_prop.integrated == 1 ) // is integrated gpu
    {
	printf(" WARNING: integrated device found for device: %i ! \n", dev_idx);
	printf(" this may lead to unsupported computation error...\n");
    }
    //now try to hook on to that device
    err = hipSetDevice(dev_idx);
    if (err != hipSuccess)
    {
        printf("Failed to set device %i: %u \n", dev_idx, err);
        goto Error;
    }
    while(true)
    {
        // see if this device is available
        err =  hipMemGetInfo(&freeBytes, &totalBytes);
        if (err != hipSuccess)
        {
            printf("Failed to get usage for device %i: %s\n", dev_idx, 
                hipGetErrorString(err));
            goto Error;
        }
        usedBytes = totalBytes - freeBytes;
        memusage = 100.0*usedBytes/totalBytes;
        // now get some device flags
        err = hipGetDeviceFlags(&flag);
        if (err != hipSuccess)
        {
            printf("Failed to get the device flags %i: %s\n", dev_idx, 
               hipGetErrorString(err));
        }
        // printf("Dev = %i, flag = %i \n", dev_idx, flag);
        if (flag != hipDeviceScheduleYield && memusage < 50)
        // if (memusage < 50)
        // if (flag != hipDeviceScheduleYield)
        {
            // If the number of CUDA contexts is greater than the number of 
            // logical processors in the system, use Spin scheduling. 
            // Else use Yield scheduling.
            err = hipSetDeviceFlags(hipDeviceScheduleYield);
            if (err != hipSuccess)
            {
                printf("Failed to set the device flag %i: %s\n", dev_idx, 
                  hipGetErrorString(err));
	    }
            printf(" # Dev Status  : GPU-mem = %f %%, PCI %i: %i\n", 
                   memusage, dev_prop.pciBusID, dev_prop.pciDeviceID );
            break;
        }
        // else let it spin
        sleep(0.02);
    }
    printf(" # Dev Selected: %i. %s \n", dev_idx, dev_prop.name);
    return 0;
Error:
    printf("Error: Failed to attach to device: %i \n", dev_idx);
    return 1;
}
/*
// function called from main fortran program 
extern "C" int cf_hookCtx(int dev_idx)
{
    nvmlReturn_t result;
    hipError_t err;
    hipCtx_t thisCtx;
    nvmlDevice_t device;
    nvmlPciInfo_t pci_bus;
    nvmlUtilization_t usage;
    unsigned int Ndevice;
    // unsigned int i;
    // unsigned int nProc = 32;
    // nvmlProcessInfo_t pInfo[nProc];
    char device_name[NVML_DEVICE_NAME_BUFFER_SIZE];

    // First initialize NVML library
    result = nvmlInit();
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        return 1;
    }
    result = nvmlDeviceGetCount(&Ndevice);
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        return 1;
    }
    if (dev_idx + 1 > Ndevice) 
    { 
        printf("Error: not enough devices detected \n");
        printf("required device idx = %i, Ndevice = %u \n", dev_idx, Ndevice);
        goto Error;
    }
    err = hipSetDevice(dev_idx);
	if( err != hipSuccess )
    {
        printf("Failed to set device %i: %u \n", dev_idx, err);
        goto Error;
    }
    while(true)
    {
        // Query for device handle to perform operations on a device
        // You can also query device handle by other features like:
        // nvmlDeviceGetHandleBySerial
        // nvmlDeviceGetHandleByPciBusId
        result = nvmlDeviceGetHandleByIndex(dev_idx, &device);
        if (NVML_SUCCESS != result)
        { 
            printf("Failed to get handle for device %i: %s\n", dev_idx, nvmlErrorString(result));
            goto Error;
        }
        // now get the current context (if any)
        hipCtxGetCurrent(&thisCtx);
        if(thisCtx == NULL) // first call to this device
        {
            hipCtxCreate(&thisCtx, 0, dev_idx);
            result = nvmlDeviceGetName(device, device_name, NVML_DEVICE_NAME_BUFFER_SIZE);
            if (NVML_SUCCESS != result)
            { 
                printf("Failed to get name of device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            // pci.busId is very useful to know which device physically 
            // you're talking to
            // Using PCI identifier you can also match nvmlDevice 
            // handle to CUDA device.
            result = nvmlDeviceGetPciInfo(device, &pci_bus);
            if (NVML_SUCCESS != result)
            { 
                printf("Failed to get pci info for device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            break;// just go ahead to initialize the case
        }
        else// see if this device is available
        {
            hipCtxSetCurrent(thisCtx);
            result = nvmlDeviceGetUtilizationRates( device, &usage );
            if (NVML_SUCCESS != result)
            {
                printf("Failed to get usage for device %i: %s\n", dev_idx, nvmlErrorString(result));
                goto Error;
            }
            if (usage.memory+usage.gpu < 15)
            {
                printf(" # Dev Status  : GPU-util=%u GPU-mem=%u \n", usage.gpu, usage.memory );
                result = nvmlDeviceGetName(device, device_name, NVML_DEVICE_NAME_BUFFER_SIZE);
                if (NVML_SUCCESS != result)
                { 
                    printf("Failed to get name of device %i: %s\n", dev_idx, nvmlErrorString(result));
                    goto Error;
                }
                // pci.busId is very useful to know which device physically 
                // you're talking to
                // Using PCI identifier you can also match nvmlDevice 
                // handle to CUDA device.
                result = nvmlDeviceGetPciInfo(device, &pci_bus);
                if (NVML_SUCCESS != result)
                { 
                    printf("Failed to get pci info for device %i: %s\n", dev_idx, nvmlErrorString(result));
                    goto Error;
                }
                break;
            }
        }
        sleep(0.05);
    }
    printf(" # Dev Selected:  %i. %s [%s]\n", dev_idx, device_name, pci_bus.busId);
    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    return 0;
Error:
    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));

    return 1;
}
*/

